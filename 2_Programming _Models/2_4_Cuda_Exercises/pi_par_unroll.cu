
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

// Definition of CUDA kernels
// ... 

int main(int argc, char *argv[]) {
  float t_seq, t_par, sp, ep;
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);

  // Adjust the number of rectangules
  int num_steps = 100000;
  if (argc == 2) {
    num_steps = atoi(argv[1]);
  } else if (argc > 2) {
    printf("Wrong number of parameters\n");
    printf("./a.out [ num_steps ]\n");
    exit(-1);
  }

/*************************************/
/******** Computation of pi **********/
/*************************************/

  int i;
  double step = 1.0 / (double) num_steps;  
  double pi = 0.0;

  //
  // Sequential implementation
  //
  double x, sum = 0.0;
  hipEventRecord(start);
  step = 1.0 / (double) num_steps;  
  for (i=0; i<num_steps; i++){
     x = (i+0.5)*step;
     sum = sum + 4.0/(1.0+x*x);
  }
  pi = step * sum;
  hipEventRecord(stop);
  hipEventSynchronize(stop);
  t_seq = 0.0;
  hipEventElapsedTime(&t_seq, start, stop);

  printf(" pi_seq = %20.15f\n", pi);
  printf(" time_seq = %20.15f\n", t_seq);

  //
  // Parallel implementation
  //
  
  // Defining the number of active threads
  int size = 0;
  
  hipEventRecord(start);
  // Call to the CUDA 
  // ...
  hipEventRecord(stop);
  hipEventSynchronize(stop);
  t_par = 0.0; 
  hipEventElapsedTime(&t_par, start, stop);
  sp = t_seq / t_par;
  ep = sp / size;

  printf(" pi_par = %20.15f\n", pi);
  printf(" time_par = %20.15f, Sp = %20.15f , Ep = %20.15f\n", t_par, sp, ep);
}
