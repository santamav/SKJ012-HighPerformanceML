#include <stdio.h> 
#include <hip/hip_runtime.h>

__global__ void mykernel( void ) {
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  printf("Hello world from! %d\n", index);
}

int main( int argc, char *argv[] ) {  
  mykernel <<<2,2>>> ();
  hipDeviceSynchronize();

  return 0;
}

