#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

// Definition of CUDA kernels
// Atomic add for double precision otherwise could not compile on Patan
__device__ double atomicAddDouble(double* address, double val)
{
    unsigned long long int* address_as_ull =
                              (unsigned long long int*)address;
    unsigned long long int old = *address_as_ull, assumed;
    do {
        assumed = old;
        old = atomicCAS(address_as_ull, assumed,
                        __double_as_longlong(val +
                               __longlong_as_double(assumed)));
    } while (assumed != old);
    return __longlong_as_double(old);
}

// Pi accumulation kernel
__global__ void accumulate_pi(double* sum, int num_steps, double step){
  int index = threadIdx.x + blockIdx.x * blockDim.x;

  int window = num_steps / (gridDim.x * blockDim.x);
  int start = index * window;
  int end = (index + 1) * window;
  if (index == (gridDim.x * blockDim.x) -1) end = num_steps;

  double local_sum = 0.0;
  for (int i = start; i < end; i++){
    double x = (i + 0.5) * step;
    local_sum += 4.0 / (1.0 + x * x);
  }

  atomicAddDouble(sum, local_sum);
}

int main(int argc, char *argv[]) {
  float t_seq, t_par, sp, ep;
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);

  // Adjust the number of rectangules
  int num_steps = 100000;
  int num_blocks = 2;
  int num_threads = 1024;
  if (argc == 2) {
      num_steps = atoi(argv[1]);
  } else if (argc == 4) {
    num_blocks = atoi(argv[1]);
    num_threads = atoi(argv[2]);
    num_steps = atoi(argv[3]);
    printf("Using %d blocks, %d threads and %d steps\n", num_blocks, num_threads, num_steps);
  }
  else if (argc > 4) {
    printf("Wrong number of parameters\n");
    printf("./a.out [ num_steps ]\n");
    exit(-1);
  }

/*************************************/
/******** Computation of pi **********/
/*************************************/
  int i;
  double step = 1.0 / (double) num_steps;  
  double pi = 0.0;

  //
  // Sequential implementation
  //
  double x, sum = 0.0;
  hipEventRecord(start);
  step = 1.0 / (double) num_steps;  
  for (i=0; i<num_steps; i++){
    x = (i+0.5)*step;
    sum = sum + 4.0/(1.0+x*x);
  }
  pi = step * sum;
  hipEventRecord(stop);
  hipEventSynchronize(stop);
  t_seq = 0.0;
  hipEventElapsedTime(&t_seq, start, stop);

  printf(" pi_seq = %20.15f\n", pi);
  printf(" time_seq = %20.15f\n", t_seq);

  //
  // Parallel implementation
  //
  
  // Defining the number of active threads
  int size = num_blocks * num_threads;
  sum = 0.0;
  hipEventRecord(start);
  // Call to the CUDA 
  //Allocate memory on the device
  double *d_sum;
  hipMalloc((void**)&d_sum, sizeof(double));
  //Initialize the device memory
  hipMemset(d_sum, 0, sizeof(double));
  //Launch the kernel
  accumulate_pi<<<num_blocks, num_threads>>>(d_sum, num_steps, step);
  //Copy the result from device to host
  hipMemcpy(&sum, d_sum, sizeof(double), hipMemcpyDeviceToHost);
  //Free the device memory
  hipFree(d_sum);
  pi = step * sum;
  hipEventRecord(stop);
  hipEventSynchronize(stop);
  t_par = 0.0;
  hipEventElapsedTime(&t_par, start, stop);
  sp = t_seq / t_par;
  ep = sp / size;

  printf(" pi_par = %20.15f\n", pi);
  printf(" time_par = %20.15f, Sp = %20.15f , Ep = %20.15f\n", t_par, sp, ep);
}
